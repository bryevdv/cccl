//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/buffer>
#include <cuda/experimental/memory_resource>
#include <cuda/std/cassert>
#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/std/utility>
#include <cuda/stream_ref>

#include <catch2/catch.hpp>

struct do_not_construct
{
  do_not_construct()
  {
    CHECK(false);
  }
};

struct my_property
{
  using value_type = int;
};
constexpr int get_property(const cuda::experimental::uninitialized_async_buffer<int, my_property>&, my_property)
{
  return 42;
}

TEMPLATE_TEST_CASE(
  "uninitialized_async_buffer", "[container]", char, short, int, long, long long, float, double, do_not_construct)
{
  using uninitialized_async_buffer = cuda::experimental::uninitialized_async_buffer<TestType>;
  static_assert(!cuda::std::is_default_constructible<uninitialized_async_buffer>::value, "");
  static_assert(!cuda::std::is_copy_constructible<uninitialized_async_buffer>::value, "");
  static_assert(!cuda::std::is_copy_assignable<uninitialized_async_buffer>::value, "");

  cuda::experimental::mr::cuda_async_memory_resource resource{};

  hipStream_t raw_stream;
  hipStreamCreate(&raw_stream);
  cuda::stream_ref stream{raw_stream};

  SECTION("construction")
  {
    {
      uninitialized_async_buffer from_stream_count{resource, stream, 42};
      CHECK(from_stream_count.data() != nullptr);
      CHECK(from_stream_count.size() == 42);
    }
    {
      uninitialized_async_buffer input{resource, stream, 42};
      const TestType* ptr = input.data();

      uninitialized_async_buffer from_rvalue{cuda::std::move(input)};
      CHECK(from_rvalue.data() == ptr);
      CHECK(from_rvalue.size() == 42);
      CHECK(from_rvalue.stream() == stream);

      // Ensure that we properly reset the input buffer
      CHECK(input.data() == nullptr);
      CHECK(input.size() == 0);
      CHECK(input.stream() == cuda::stream_ref{});
    }

    hipStream_t other_raw_stream;
    hipStreamCreate(&other_raw_stream);
    cuda::stream_ref other_stream{other_raw_stream};
    {
      uninitialized_async_buffer input{resource, other_stream, 42};
      const TestType* ptr = input.data();

      uninitialized_async_buffer assign_rvalue{resource, stream, 1337};
      assign_rvalue = cuda::std::move(input);
      CHECK(assign_rvalue.data() == ptr);
      CHECK(assign_rvalue.size() == 42);
      CHECK(assign_rvalue.stream() == other_stream);

      // Ensure that we properly reset the input buffer
      CHECK(input.data() == nullptr);
      CHECK(input.size() == 0);
      CHECK(input.stream() == cuda::stream_ref{});
    }
    hipStreamDestroy(other_raw_stream);
  }

  SECTION("access")
  {
    uninitialized_async_buffer buf{resource, stream, 42};
    CHECK(buf.data() != nullptr);
    CHECK(buf.size() == 42);
    CHECK(buf.begin() == buf.data());
    CHECK(buf.end() == buf.begin() + buf.size());
    CHECK(buf.stream() == stream);

    CHECK(cuda::std::as_const(buf).data() != nullptr);
    CHECK(cuda::std::as_const(buf).size() == 42);
    CHECK(cuda::std::as_const(buf).begin() == buf.data());
    CHECK(cuda::std::as_const(buf).end() == buf.begin() + buf.size());
    CHECK(cuda::std::as_const(buf).stream() == stream);
  }

  SECTION("properties")
  {
    static_assert(cuda::has_property<cuda::experimental::uninitialized_async_buffer<int, cuda::mr::device_accessible>,
                                     cuda::mr::device_accessible>,
                  "");
    static_assert(cuda::has_property<cuda::experimental::uninitialized_async_buffer<int, my_property>, my_property>,
                  "");
  }

  SECTION("convertion to span")
  {
    uninitialized_async_buffer buf{resource, stream, 42};
    const cuda::std::span<TestType> as_span{buf};
    CHECK(as_span.data() == buf.data());
    CHECK(as_span.size() == 42);
  }

  hipStreamDestroy(raw_stream);
}
